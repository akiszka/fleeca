#include "hip/hip_runtime.h"
#include <CImg.h>
#include <iostream>
#include <thrust/complex.h>

using namespace cimg_library;

using u = std::uint32_t;
using i = std::int32_t;

constexpr u max_iter = 2000;

constexpr u window_x = 1920*4;
constexpr u window_y = 1080*4;

__global__ void generate_set(u win_x, u win_y, u* result, u max_iter);
__device__ double map(double x, double in_min, double in_max, double out_min, double out_max);
__device__ u generate_color(std::uint8_t r, std::uint8_t g, std::uint8_t b);
__device__ u iter_to_color(u iterations, u max_iter);

int main() {
    CImg<unsigned char> theImage(window_x, window_y, 1, 3, 0);

    u* res_arr;
    hipMallocManaged(&res_arr, sizeof(u)*window_x*window_y*2 /* FIXME: to avoid segfaults I added a *2 to the allocated memory */);

    generate_set<<<24,512>>>(window_x, window_y, res_arr, max_iter);
    hipDeviceSynchronize();

    for (u x = 0; x < window_x; ++x) {
	for (u y = 0; y < window_y; ++y) {
	    u color = res_arr[x*window_x+y];
	    theImage(x, y, 0) =  color & 0xff;
	    theImage(x, y, 1) = (color>>8) & 0xff;
	    theImage(x, y, 2) = (color>>16) & 0xff;
	}
    }

    hipFree(res_arr);

    theImage.save("img.png");

    return 0;
}

__global__ void generate_set(u win_x, u win_y, u* result, u max_iter) {
    const thrust::complex<double> julia_c(-0.1f, 0.65f);

    u iter = 0;

    u x_index = blockIdx.x;
    u x_stride = gridDim.x;

    u y_index = threadIdx.x;
    u y_stride = blockDim.x;

    for (u x = x_index; x < win_x; x += x_stride) {
	for(u y = y_index; y < win_y; y += y_stride) {
	    thrust::complex<double> a (
		map(x, 0.0f, win_x, -2.5f, 1.5f),
		map(y, 0.0f, win_y, -1.25f, 1.25f)
		);
	    thrust::complex<double> z = 0;

	    for (iter = 0; abs(z) <= 2 && iter < max_iter; ++iter) {
		z = z*z + a;
	    }

	    result[x*window_x+y] = iter_to_color(iter, max_iter);
	}
    }
}

__device__ u generate_color(std::uint8_t r, std::uint8_t g, std::uint8_t b) {
    return (b<<16) + (g<<8) + r;
}

__device__ u iter_to_color(u iterations, u max_iter) {
    switch (iterations*7/max_iter) {
    case 0:
	return generate_color(255, 0, 0); // red
	break;
    case 1:
    case 2:
	return generate_color(255, 128, 0); // orange
	break;
    case 3:
	return generate_color(0, 255, 0); // green
	break;
    case 4:
	return generate_color(0, 255, 128); // blue-greenish
	break;
    case 5:
	return generate_color(0, 255, 255); // light blue
	break;
    case 6:
	return generate_color(0, 0, 255); // dark blue
	break;
    case 7:
	return generate_color(0, 0, 0);
	break;
    }

    return 0;
}

__device__ double map(double x, double in_min, double in_max, double out_min, double out_max) {
    return (x - in_min) * (out_max - out_min) / (in_max - in_min) + out_min;
}